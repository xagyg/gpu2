#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <time.h>
#include <math.h>
#include <memory.h>


const int numPeople = 5000;
const int loopCount = 10000000;  // Simulated heavy computation

__global__ void computePay(float* hours, float* rate, float* pay, int numPeople) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numPeople) {
        float p = hours[idx] * rate[idx];
        for (int j = 0; j < loopCount; j++) {
            p *= 1.0000001f;
            p /= 1.0000001f;
        }
        pay[idx] = p;
    }
}

int main() {
    float* h_hoursWorked = (float*)malloc(numPeople * sizeof(float));
    float* h_hourlyRate = (float*)malloc(numPeople * sizeof(float));
    float* h_pay = (float*)malloc(numPeople * sizeof(float));

    // Fill with random values
    srand((unsigned int)time(NULL));
    for (int i = 0; i < numPeople; i++) {
        h_hoursWorked[i] = 16 + rand() % 32;               // 16–47 hours
        h_hourlyRate[i] = 15 + rand() % 16;               // $15–$30
    }

    clock_t start, end;
    double cpu_time_used;
    start = clock();  // Start timing

    // Allocate device memory
    float* d_hoursWorked, * d_hourlyRate, * d_pay;
    hipMalloc(&d_hoursWorked, numPeople * sizeof(float));
    hipMalloc(&d_hourlyRate, numPeople * sizeof(float));
    hipMalloc(&d_pay, numPeople * sizeof(float));

    // Copy input to device
    hipMemcpy(d_hoursWorked, h_hoursWorked, numPeople * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_hourlyRate, h_hourlyRate, numPeople * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numPeople + threadsPerBlock - 1) / threadsPerBlock;

    //hipEvent_t start, stop;
    //hipEventCreate(&start);
    //hipEventCreate(&stop);
    //hipEventRecord(start);

    computePay << <blocksPerGrid, threadsPerBlock >> > (d_hoursWorked, d_hourlyRate, d_pay, numPeople);

    //hipEventRecord(stop);
    //hipEventSynchronize(stop);

    //float milliseconds = 0;
   // hipEventElapsedTime(&milliseconds, start, stop);
    //printf("GPU time: %.3f seconds\n", milliseconds / 1000.0f);

    // Copy result back
    hipMemcpy(h_pay, d_pay, numPeople * sizeof(float), hipMemcpyDeviceToHost);

    end = clock();  // End timing
    cpu_time_used = ((double)(end - start)) / CLOCKS_PER_SEC;

    // Print the result
    printf("Pay for each person:\n");
    for (int i = 0; i < numPeople; i++) {
        printf("Person %d: $%.2f\n", i, h_pay[i]);
    }

    printf("Time taken: %f seconds\n", cpu_time_used);

    // Clean up
    hipFree(d_hoursWorked);
    hipFree(d_hourlyRate);
    hipFree(d_pay);
    free(h_hoursWorked);
    free(h_hourlyRate);
    free(h_pay);

    return 0;
}
